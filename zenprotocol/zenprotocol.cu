#include "hip/hip_runtime.h"
/**
 * Zen Protocol SHA3.
 */

#include <miner.h>
#include <cuda_helper.h>
#include <openssl/evp.h>

static uint32_t *d_resultNonce[MAX_GPUS];

extern "C" void zenprotocol_hash(void *output, const uint32_t *input)
{

#if 0
  printf("CPU INPUT\n");
  for (int i = 0; i < 25; ++i) {
    printf("%08x ", input[i]);
  }
  printf("\n");
#endif

  uint32_t hash[8];

  EVP_MD_CTX *ctx = EVP_MD_CTX_create();
  EVP_DigestInit(ctx, EVP_sha3_256());
  EVP_DigestUpdate(ctx, (uint8_t*)input, 100);
  EVP_DigestFinal(ctx, (uint8_t*)hash, NULL);

#if 0  
  printf("############ SHA3 CPU HASH:\n");
  for (int i = 0; i < 8; ++i) {
    printf("%08x ", ((uint32_t*)hash)[i]);
  }
  printf("\n");
#endif
}


static bool init[MAX_GPUS] = { 0 };

extern void zenprotocol_init(int thr_id);
extern void zenprotocol_free(int thr_id);
extern void zenprotocol_setBlock_100(uint32_t *pdata);
extern void zenprotocol_setTarget(const uint32_t *ptarget);
extern void zenprotocol_cpu_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resultNonces);

extern "C" int scanhash_zenprotocol(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
        uint32_t *nonceptr = &work->data[24];
        const uint32_t first_nonce = *nonceptr;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 26);
	if (init[thr_id]) throughput = min(throughput, (max_nonce - *nonceptr));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x03;
        
	if (!init[thr_id])
	{
          CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		zenprotocol_init(thr_id);
                CUDA_SAFE_CALL(hipMalloc(&d_resultNonce[thr_id], 2 * sizeof(uint32_t)));

		init[thr_id] = true;
	}

	zenprotocol_setBlock_100(work->data);
        zenprotocol_setTarget(ptarget);
        work->valid_nonces = 0;
	uint32_t start_nonce = *nonceptr;

        CUDA_SAFE_CALL(hipMemset(d_resultNonce[thr_id], 0xFF, 2 * sizeof(uint32_t)));
	do {
                *hashes_done = *nonceptr - first_nonce + throughput;
                zenprotocol_cpu_hash(thr_id, throughput, start_nonce, d_resultNonce[thr_id]);
                CUDA_SAFE_CALL(hipMemcpy(&work->nonces[0], d_resultNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];
		        work->nonces[0] += start_nonce;
			work->data[24] = work->nonces[0];
			zenprotocol_hash(vhash, work->data);

                        hipMemset(d_resultNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));
			if (true) {
			  work->valid_nonces = 1;
			  work_set_target_ratio(work, vhash);
			
			  return 1;
			} else {
			  gpu_increment_reject(thr_id);
			  if (!opt_quiet)
			    gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
                          start_nonce += throughput;
			}
		}

		if ((uint64_t) throughput + *nonceptr >= max_nonce) {
			*nonceptr = max_nonce;
			break;
		}
		*nonceptr += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = *nonceptr - first_nonce;

	return 0;
}

extern "C" void free_zenprotocol(int thr_id)
{
	if (!init[thr_id])
		return;

        if (d_resultNonce[thr_id]) {
          hipFree(d_resultNonce[thr_id]);
        }
        d_resultNonce[thr_id] = NULL;
	init[thr_id] = false;
}
