#include "hip/hip_runtime.h"
/*
 * Zen Protocol SHA3 CUDA implementation.
 */

#include <stdint.h>
#include <stdio.h>
#include <memory.h>

#include <cuda_helper.h>
#include <miner.h>

__constant__ uint32_t pTarget[8];
__constant__ static uint8_t c_PaddedMessage[136]; // padded message (100 bytes + padding)



static __device__ uint32_t HIWORD(const uint64_t x)
{
    uint32_t result;
    asm(
        "{\n\t"
        ".reg .u32 xl; \n\t"
        "mov.b64 {xl,%0},%1; \n\t"
        "}" : "=r"(result) : "l"(x)
    );
    return result;
}

static __device__ uint32_t LOWORD(const uint64_t x)
{
    uint32_t result;
    asm(
        "{\n\t"
        ".reg .u32 xh; \n\t"
        "mov.b64 {%0,xh},%1; \n\t"
        "}" : "=r"(result) : "l"(x)
    );
    return result;
}

#define ROTL_1(d0, d1, v0, v1)      ROTL_SMALL(d0, d1, v0, v1,  1)
#define ROTL_2(d0, d1, v0, v1)      ROTL_SMALL(d0, d1, v0, v1,  2)
#define ROTL_3(d0, d1, v0, v1)      ROTL_SMALL(d0, d1, v0, v1,  3)
#define ROTL_6(d0, d1, v0, v1)      ROTL_SMALL(d0, d1, v0, v1,  6)
#define ROTL_8(d0, d1, v0, v1)      ROTL_SMALL(d0, d1, v0, v1,  8)
#define ROTL_10(d0, d1, v0, v1)     ROTL_SMALL(d0, d1, v0, v1, 10)
#define ROTL_14(d0, d1, v0, v1)     ROTL_SMALL(d0, d1, v0, v1, 14)
#define ROTL_15(d0, d1, v0, v1)     ROTL_SMALL(d0, d1, v0, v1, 15)
#define ROTL_18(d0, d1, v0, v1)     ROTL_SMALL(d0, d1, v0, v1, 18)
#define ROTL_20(d0, d1, v0, v1)     ROTL_SMALL(d0, d1, v0, v1, 20)
#define ROTL_21(d0, d1, v0, v1)     ROTL_SMALL(d0, d1, v0, v1, 21)
#define ROTL_25(d0, d1, v0, v1)     ROTL_SMALL(d0, d1, v0, v1, 25)
#define ROTL_27(d0, d1, v0, v1)     ROTL_SMALL(d0, d1, v0, v1, 27)
#define ROTL_28(d0, d1, v0, v1)     ROTL_SMALL(d0, d1, v0, v1, 28)
#define ROTL_32(d0, d1, v0, v1)     (d0 = v1; d1 = v0; )
#define ROTL_36(d0, d1, v0, v1)     ROTL_BIG(d0, d1, v0, v1, 36)
#define ROTL_39(d0, d1, v0, v1)     ROTL_BIG(d0, d1, v0, v1, 39)
#define ROTL_41(d0, d1, v0, v1)     ROTL_BIG(d0, d1, v0, v1, 41)
#define ROTL_43(d0, d1, v0, v1)     ROTL_BIG(d0, d1, v0, v1, 43)
#define ROTL_44(d0, d1, v0, v1)     ROTL_BIG(d0, d1, v0, v1, 44)
#define ROTL_45(d0, d1, v0, v1)     ROTL_BIG(d0, d1, v0, v1, 45)
#define ROTL_55(d0, d1, v0, v1)     ROTL_BIG(d0, d1, v0, v1, 55)
#define ROTL_56(d0, d1, v0, v1)     ROTL_BIG(d0, d1, v0, v1, 56)
#define ROTL_61(d0, d1, v0, v1)     ROTL_BIG(d0, d1, v0, v1, 61)
#define ROTL_62(d0, d1, v0, v1)     ROTL_BIG(d0, d1, v0, v1, 62)

#define ROTLI_1(d1, d2, v1, v2)    ROTLI_odd1(d1, d2, v1, v2)
#define ROTLI_2(d1, d2, v1, v2)    ROTLI_even(d1, d2, v1, v2,  1)
#define ROTLI_3(d1, d2, v1, v2)    ROTLI_odd( d1, d2, v1, v2,  2)
#define ROTLI_6(d1, d2, v1, v2)    ROTLI_even(d1, d2, v1, v2,  3)
#define ROTLI_8(d1, d2, v1, v2)    ROTLI_even(d1, d2, v1, v2,  4)
#define ROTLI_10(d1, d2, v1, v2)   ROTLI_even(d1, d2, v1, v2,  5)
#define ROTLI_14(d1, d2, v1, v2)   ROTLI_even(d1, d2, v1, v2,  7)
#define ROTLI_15(d1, d2, v1, v2)   ROTLI_odd( d1, d2, v1, v2,  8)
#define ROTLI_18(d1, d2, v1, v2)   ROTLI_even(d1, d2, v1, v2,  9)
#define ROTLI_20(d1, d2, v1, v2)   ROTLI_even(d1, d2, v1, v2, 10)
#define ROTLI_21(d1, d2, v1, v2)   ROTLI_odd( d1, d2, v1, v2, 11)
#define ROTLI_25(d1, d2, v1, v2)   ROTLI_odd( d1, d2, v1, v2, 13)
#define ROTLI_27(d1, d2, v1, v2)   ROTLI_odd( d1, d2, v1, v2, 14)
#define ROTLI_28(d1, d2, v1, v2)   ROTLI_even(d1, d2, v1, v2, 14)
#define ROTLI_36(d1, d2, v1, v2)   ROTLI_even(d1, d2, v1, v2, 18)
#define ROTLI_39(d1, d2, v1, v2)   ROTLI_odd( d1, d2, v1, v2, 20)
#define ROTLI_41(d1, d2, v1, v2)   ROTLI_odd( d1, d2, v1, v2, 21)
#define ROTLI_43(d1, d2, v1, v2)   ROTLI_odd( d1, d2, v1, v2, 22)
#define ROTLI_44(d1, d2, v1, v2)   ROTLI_even(d1, d2, v1, v2, 22)
#define ROTLI_45(d1, d2, v1, v2)   ROTLI_odd( d1, d2, v1, v2, 23)
#define ROTLI_55(d1, d2, v1, v2)   ROTLI_odd( d1, d2, v1, v2, 28)
#define ROTLI_56(d1, d2, v1, v2)   ROTLI_even(d1, d2, v1, v2, 28)
#define ROTLI_61(d1, d2, v1, v2)   ROTLI_odd( d1, d2, v1, v2, 31)
#define ROTLI_62(d1, d2, v1, v2)   ROTLI_even(d1, d2, v1, v2, 31)

#define ROTs(a, b, n) ROTL_##n(s[a], s[a+1], s[b], s[b+1])
#define ROTIs(a, b, n) ROTLI_##n(s[a], s[a+1], s[b], s[b+1])

static __device__ __forceinline__ void ROTL_SMALL( uint32_t &d0, uint32_t &d1, uint32_t v0, uint32_t v1, const uint32_t offset )
{
#if __CUDA_ARCH__ >= 320
    asm(
        "shf.l.wrap.b32 %0, %2, %3, %4;\n\t"
        "shf.l.wrap.b32 %1, %3, %2, %4;\n\t"
        : "=r"(d0), "=r"(d1) 
        : "r"(v1), "r"(v0), "r"(offset));
#else
    d0 = (v0 << offset) | (v1 >> (32-offset));
    d1 = (v1 << offset) | (v0 >> (32-offset));
#endif
}

static __device__ __forceinline__ void ROTL_BIG( uint32_t &d0, uint32_t &d1, uint32_t v0, uint32_t v1, const uint32_t offset )
{
#if __CUDA_ARCH__ >= 320
    asm(
        "shf.l.wrap.b32 %0, %3, %2, %4;\n\t"
        "shf.l.wrap.b32 %1, %2, %3, %4;\n\t"
        : "=r"(d0), "=r"(d1) 
        : "r"(v1), "r"(v0), "r"(offset-32));
#else
    d0 = (v1 << (offset-32)) | (v0 >> (64-offset));
    d1 = (v0 << (offset-32)) | (v1 >> (64-offset));
#endif
}

__constant__ uint32_t d_RC[48];
static const uint32_t h_RC[48] = {
    0x00000001, 0x00000000, 0x00008082, 0x00000000,
    0x0000808a, 0x80000000, 0x80008000, 0x80000000,
    0x0000808b, 0x00000000, 0x80000001, 0x00000000,
    0x80008081, 0x80000000, 0x00008009, 0x80000000,
    0x0000008a, 0x00000000, 0x00000088, 0x00000000,
    0x80008009, 0x00000000, 0x8000000a, 0x00000000,
    0x8000808b, 0x00000000, 0x0000008b, 0x80000000,
    0x00008089, 0x80000000, 0x00008003, 0x80000000,
    0x00008002, 0x80000000, 0x00000080, 0x80000000,
    0x0000800a, 0x00000000, 0x8000000a, 0x80000000,
    0x80008081, 0x80000000, 0x00008080, 0x80000000,
    0x80000001, 0x00000000, 0x80008008, 0x80000000
};

static __device__ void keccak_block(uint32_t *s) 
{
    uint32_t t[10], u[10], v[2];

#pragma unroll 4
    for (int i = 0; i < 48; i += 2) {

        t[4] = s[4] ^ s[14] ^ s[24] ^ s[34] ^ s[44];
        t[5] = s[5] ^ s[15] ^ s[25] ^ s[35] ^ s[45];
        t[2] = s[2] ^ s[12] ^ s[22] ^ s[32] ^ s[42];
        t[3] = s[3] ^ s[13] ^ s[23] ^ s[33] ^ s[43];
        t[6] = s[6] ^ s[16] ^ s[26] ^ s[36] ^ s[46];
        t[7] = s[7] ^ s[17] ^ s[27] ^ s[37] ^ s[47];
        t[8] = s[8] ^ s[18] ^ s[28] ^ s[38] ^ s[48];
        t[9] = s[9] ^ s[19] ^ s[29] ^ s[39] ^ s[49];
        t[0] = s[0] ^ s[10] ^ s[20] ^ s[30] ^ s[40];
        t[1] = s[1] ^ s[11] ^ s[21] ^ s[31] ^ s[41];
    
        ROTL_1(u[2], u[3], t[4], t[5]);
        ROTL_1(u[0], u[1], t[2], t[3]);
        ROTL_1(u[4], u[5], t[6], t[7]);
        ROTL_1(u[6], u[7], t[8], t[9]);
        ROTL_1(u[8], u[9], t[0], t[1]);
        
        u[2] ^= t[0]; u[3] ^= t[1];
        u[0] ^= t[8]; u[1] ^= t[9];
        u[4] ^= t[2]; u[5] ^= t[3];
        u[6] ^= t[4]; u[7] ^= t[5];
        u[8] ^= t[6]; u[9] ^= t[7];

        s[2] ^= u[2]; s[3] ^= u[3];

        s[0] ^= u[0]; s[10] ^= u[0]; s[20] ^= u[0]; s[30] ^= u[0]; s[40] ^= u[0];
        s[1] ^= u[1]; s[11] ^= u[1]; s[21] ^= u[1]; s[31] ^= u[1]; s[41] ^= u[1];
        s[12] ^= u[2]; s[22] ^= u[2]; s[32] ^= u[2]; s[42] ^= u[2];
        s[13] ^= u[3]; s[23] ^= u[3]; s[33] ^= u[3]; s[43] ^= u[3];
        s[4] ^= u[4]; s[14] ^= u[4]; s[24] ^= u[4]; s[34] ^= u[4]; s[44] ^= u[4];
        s[5] ^= u[5]; s[15] ^= u[5]; s[25] ^= u[5]; s[35] ^= u[5]; s[45] ^= u[5];
        s[6] ^= u[6]; s[16] ^= u[6]; s[26] ^= u[6]; s[36] ^= u[6]; s[46] ^= u[6];
        s[7] ^= u[7]; s[17] ^= u[7]; s[27] ^= u[7]; s[37] ^= u[7]; s[47] ^= u[7];
        s[8] ^= u[8]; s[18] ^= u[8]; s[28] ^= u[8]; s[38] ^= u[8]; s[48] ^= u[8];
        s[9] ^= u[9]; s[19] ^= u[9]; s[29] ^= u[9]; s[39] ^= u[9]; s[49] ^= u[9];

        v[0] = s[2]; v[1] = s[3];
        ROTs( 2, 12, 44);
        ROTs(12, 18, 20);
        ROTs(18, 44, 61);
        ROTs(44, 28, 39);
        ROTs(28, 40, 18);
        ROTs(40,  4, 62);
        ROTs( 4, 24, 43);
        ROTs(24, 26, 25);
        ROTs(26, 38,  8);
        ROTs(38, 46, 56);
        ROTs(46, 30, 41);
        ROTs(30,  8, 27);
        ROTs( 8, 48, 14);
        ROTs(48, 42,  2);
        ROTs(42, 16, 55);
        ROTs(16, 32, 45);
        ROTs(32, 10, 36);
        ROTs(10,  6, 28);
        ROTs( 6, 36, 21);
        ROTs(36, 34, 15);
        ROTs(34, 22, 10);
        ROTs(22, 14,  6);
        ROTs(14, 20,  3);
        ROTL_1(s[20], s[21], v[0], v[1]);

        v[0] = s[ 0]; v[1] = s[ 2]; s[ 0] ^= (~v[1]) & s[ 4]; s[ 2] ^= (~s[ 4]) & s[ 6]; s[ 4] ^= (~s[ 6]) & s[ 8]; s[ 6] ^= (~s[ 8]) & v[0]; s[ 8] ^= (~v[0]) & v[1];
        v[0] = s[ 1]; v[1] = s[ 3]; s[ 1] ^= (~v[1]) & s[ 5]; s[ 3] ^= (~s[ 5]) & s[ 7]; s[ 5] ^= (~s[ 7]) & s[ 9]; s[ 7] ^= (~s[ 9]) & v[0]; s[ 9] ^= (~v[0]) & v[1];
        v[0] = s[10]; v[1] = s[12]; s[10] ^= (~v[1]) & s[14]; s[12] ^= (~s[14]) & s[16]; s[14] ^= (~s[16]) & s[18]; s[16] ^= (~s[18]) & v[0]; s[18] ^= (~v[0]) & v[1];
        v[0] = s[11]; v[1] = s[13]; s[11] ^= (~v[1]) & s[15]; s[13] ^= (~s[15]) & s[17]; s[15] ^= (~s[17]) & s[19]; s[17] ^= (~s[19]) & v[0]; s[19] ^= (~v[0]) & v[1];
        v[0] = s[20]; v[1] = s[22]; s[20] ^= (~v[1]) & s[24]; s[22] ^= (~s[24]) & s[26]; s[24] ^= (~s[26]) & s[28]; s[26] ^= (~s[28]) & v[0]; s[28] ^= (~v[0]) & v[1];
        v[0] = s[21]; v[1] = s[23]; s[21] ^= (~v[1]) & s[25]; s[23] ^= (~s[25]) & s[27]; s[25] ^= (~s[27]) & s[29]; s[27] ^= (~s[29]) & v[0]; s[29] ^= (~v[0]) & v[1];
        v[0] = s[30]; v[1] = s[32]; s[30] ^= (~v[1]) & s[34]; s[32] ^= (~s[34]) & s[36]; s[34] ^= (~s[36]) & s[38]; s[36] ^= (~s[38]) & v[0]; s[38] ^= (~v[0]) & v[1];
        v[0] = s[31]; v[1] = s[33]; s[31] ^= (~v[1]) & s[35]; s[33] ^= (~s[35]) & s[37]; s[35] ^= (~s[37]) & s[39]; s[37] ^= (~s[39]) & v[0]; s[39] ^= (~v[0]) & v[1];
        v[0] = s[40]; v[1] = s[42]; s[40] ^= (~v[1]) & s[44]; s[42] ^= (~s[44]) & s[46]; s[44] ^= (~s[46]) & s[48]; s[46] ^= (~s[48]) & v[0]; s[48] ^= (~v[0]) & v[1];
        v[0] = s[41]; v[1] = s[43]; s[41] ^= (~v[1]) & s[45]; s[43] ^= (~s[45]) & s[47]; s[45] ^= (~s[47]) & s[49]; s[47] ^= (~s[49]) & v[0]; s[49] ^= (~v[0]) & v[1];

        s[0] ^= d_RC[i];
        s[1] ^= d_RC[i+1];
    }
}

__global__ void zenprotocol_gpu_hash_100(uint32_t threads, uint32_t startNonce, uint32_t *resultNonce)
{
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    
    if (thread < threads)
    {
        uint32_t nonce =  startNonce + thread;
        uint64_t state[25];

#pragma unroll 17
        for (int i = 0; i < 17; ++i) {
          state[i] = (((uint64_t*)c_PaddedMessage)[i]);
        }
	((uint32_t*)state)[24] = nonce;
#if 0        
        printf("GPU INPUT 0\n");
        for (int i = 0; i < 17; ++i) {
          printf("%08x", cuda_swab32(((uint32_t*)state)[i]));
        }
        printf("\n");
#endif
        
#pragma unroll 8        
        for (int i = 17; i < 25; ++i) {
          state[i] = 0;
        }

        keccak_block((uint32_t*)state);

        uint32_t h0 = cuda_swab32(LOWORD(state[0]));
        uint32_t h1 = cuda_swab32(HIWORD(state[0]));

        if ((h0 < pTarget[0]) || (h0 == pTarget[0] && h1 <= pTarget[1])) {
          uint32_t tmp = atomicExch(&resultNonce[0], thread);
          if (tmp != UINT32_MAX) {
            resultNonce[1] = tmp;
          }

#if 0
          uint32_t pHash[8];
#pragma unroll 4
          for (int i = 0; i < 4; ++i) {
            pHash[2 * i] = cuda_swab32(LOWORD(state[i]));
            pHash[2 * i + 1] = cuda_swab32(HIWORD(state[i]));
          }
          printf("GPU 100 HASH:\n");
          for (int i = 0; i < 8; ++i) {
            printf("%08x", pHash[i]);
          }
          printf("\nGPU 100 Target:\n");
          for (int i = 0; i < 8; ++i) {
            printf("%08x", pTarget[i]);
          }
	  printf("\n");
#endif
         }

    }
}

__host__ void zenprotocol_setBlock_100(uint32_t *pdata)
{
	unsigned char PaddedMessage[136];
	memcpy(PaddedMessage, pdata, 100);
	memset(PaddedMessage + 100, 0, 36);
	PaddedMessage[100] = 0x06;
	PaddedMessage[135] = 0x80;

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage), PaddedMessage, 136*sizeof(uint8_t), 0, hipMemcpyHostToDevice));
}

__host__ void zenprotocol_setTarget(const uint32_t *ptarget)
{
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ptarget, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}

__host__
void zenprotocol_init(int thr_id)
{
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_RC), h_RC, sizeof(h_RC), 0, hipMemcpyHostToDevice));
}

__host__
void zenprotocol_free(int thr_id)
{
}

__host__
void zenprotocol_cpu_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resultNonce)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

        zenprotocol_gpu_hash_100 <<<grid, block>>> (threads, startNonce, resultNonce);
}

